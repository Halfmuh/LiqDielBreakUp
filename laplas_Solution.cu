#include "hip/hip_runtime.h"
#include "laplas_Solution.cuh"
#include <FL/Fl.H>
#include <fstream>

__global__ void setDouble1(double* target,double val){
	target[threadIdx.x+threadIdx.y*blockDim.x]=val;
}

__global__ void setDouble2(double* target,double val){
	target[threadIdx.x+threadIdx.y*blockDim.x+blockIdx.x*blockDim.x*blockDim.y]=/*(double)(threadIdx.x+threadIdx.y*blockDim.x+blockIdx.x*blockDim.x*blockDim.y)/(10*10*10)*/val;
}

///���� � ���������� ���������
__global__ void sliceKernel(double* A,double*result,const int z){
	int dY=blockDim.x;
	int dZ=(gridDim.x)*(blockDim.x);

	int i=threadIdx.x+blockIdx.x*dZ;
	double a=A[i+z*dY];

	result[threadIdx.x+blockIdx.x*dY]=a;
}

__global__ void devGetFiCentral(const double* fi, double* result, int xy_idx, int dZ){

	int idxZ =(1+threadIdx.x+threadIdx.y*4+threadIdx.z*16 +blockIdx.x*32);
	int idx=xy_idx +dZ*idxZ;
	result[idxZ]=fi[idx];

}

__global__ void laplasKernel(const double* fcu, double* fncu/*, double*sig,const double*Q*/){
	int dY;
	dY=gridDim.x*32+2;
	int dZ;
	dZ=(gridDim.x*32+2)*(gridDim.y+2);
	int i0;
	i0=1+(threadIdx.x+threadIdx.y*4+threadIdx.z*16 +blockIdx.x*32) +(1+blockIdx.y)*dY+ (1+blockIdx.z)*dZ;
	//1+blockIdx.x +(blockIdx.y +1)*dY+(threadIdx.x +1)*dZ;
	//pts=4*M_PI;
	double upper_part;
	upper_part=0;
	upper_part+=fcu[i0+1];
	upper_part+=fcu[i0-1];
	upper_part+=fcu[i0+dY];
	upper_part+=fcu[i0-dY];
	upper_part+=fcu[i0+dZ];
	upper_part+=fcu[i0-dZ];
	upper_part*=1.0/6.0;

	fncu[i0]=upper_part;
	__syncthreads();
}
__global__ void addQOld(const double* Q, double* fiNew){
	int dY;
	dY=gridDim.x*32+2;
	int dZ;
	dZ=(gridDim.x*32+2)*(gridDim.y+2);
	int i0;
	i0=1+(threadIdx.x+threadIdx.y*4+threadIdx.z*16 +blockIdx.x*32) +(1+blockIdx.y)*dY+ (1+blockIdx.z)*dZ;
	fiNew[i0]+=Q[i0]*4*M_PI;
}
__global__ void addSummFi(const int* structure,double* fiNew,double* fiOld,double sigma){
	double c=2*M_PI;
	int dY;
	dY=gridDim.x*32+2;
	int dZ;
	dZ=(gridDim.x*32+2)*(gridDim.y+2);
	int i0;
	i0=1+(threadIdx.x+threadIdx.y*4+threadIdx.z*16 +blockIdx.x*32) +(1+blockIdx.y)*dY+ (1+blockIdx.z)*dZ;
	for (int i=-1;i<2;i++){
		for(int j=-1;j<2;j++){
			for(int k=-1;k<2;k++){
				int from= i0+i+j*dY+k*dZ;
				if(i0!=from){
					int tmpDirCheck= int(structure[i0]<60) *int(structure[i0]>0) *(int(structure[from]<60)*int(structure[from]>0)+int(structure[from]==101));
					fiNew[i0]+=fiOld[from]*tmpDirCheck*sigma*c/sqrt(i*i+j*j+k*k);
				}
			}
		}
	}
}
__global__ void lowerPart(const int* structure,double* fiNew, double sigma){
	double c=2*M_PI;
	double lower_part=1;
	int dY;
	dY=gridDim.x*32+2;
	int dZ;
	dZ=(gridDim.x*32+2)*(gridDim.y+2);
	int i0;
	i0=1+(threadIdx.x+threadIdx.y*4+threadIdx.z*16 +blockIdx.x*32) +(1+blockIdx.y)*dY+ (1+blockIdx.z)*dZ;
	for (int i=-1;i<2;i++){
		for(int j=-1;j<2;j++){
			for(int k=-1;k<2;k++){
				int from= i0+i+j*dY+k*dZ;
				if(i0!=from){
					int tmpDirCheck= int(structure[i0]<60) *int(structure[i0]>0) *(int(structure[from]<60)*int(structure[from]>0)+int(structure[from]==101));
					lower_part+=tmpDirCheck*sigma*c/sqrt(i*i+j*j+k*k);
				}
			}
		}
	}
	fiNew[i0]/=lower_part;
}
//__global__ void chargeKernel(
__global__ void state_field_update(int* d_states,double* d_field_target){
	int dY=gridDim.x+2;
	int dZ=(gridDim.x+2)*(gridDim.y+2);
	int i0;
	i0=1+blockIdx.x +(blockIdx.y +1)*dY+(threadIdx.x +1)*dZ;
	if((d_states[i0]<60)&&(d_states[i0]>0)) d_field_target[i0]=1;
}

__global__ void yx_Borders(double* target){
	int dY=gridDim.x;
	int dZ=gridDim.x*blockDim.x;

	int idx=threadIdx.x+(blockIdx.x*dY);
	target[idx]=1;

	idx=threadIdx.x+(blockIdx.x*dY)+dZ*(dY-1);
	target[idx]=0;
}

__global__ void update_border(double* target){

	int dY=gridDim.x*32+2;
	int dZ=(2+gridDim.x*32)*(2+gridDim.z);

	int idx=1+threadIdx.x+threadIdx.y*4+threadIdx.z*16+blockIdx.x*32+(1+blockIdx.z)*dZ; ///index of  y=const border
	target[idx]=target[idx+dY];/*
							   (double)(gridDim.x-blockIdx.x)/gridDim.x;*/
	__syncthreads();
	idx+=dZ-dY;
	target[idx]=target[idx-dY];/*
							   (double)(gridDim.x-blockIdx.x)/gridDim.x;*/
	__syncthreads();


	idx=(1+threadIdx.x+threadIdx.y*4+threadIdx.z*16+blockIdx.x*32)*dY+(1+blockIdx.z)*dZ;//x=const (0) border
	target[idx]=target[idx+1];/*
							  (double)(gridDim.x-blockIdx.x)/gridDim.x;*/
	__syncthreads();
	idx+=dY-1;// vtoraya x=const granica
	target[idx]=target[idx-1];/*
							  (double)(gridDim.x-blockIdx.x)/gridDim.x;*/
	__syncthreads();

}

__global__ void edge_update(double* target,int yx_idx,int dZ){
	int idx=yx_idx+(threadIdx.x)*dZ;   ///lesvie, stir'ek, igla ,elektrod
	target[idx]=1;
}

//// ABSOLUTE CONVERGENCE
__global__ void niv_stage_dim3(char* result,double* field_old,double* field_new, double eps){
	__syncthreads();
	int dY=gridDim.x;
	int dZ=gridDim.x*blockDim.x;
	int idx=threadIdx.x+blockIdx.x*dY+blockIdx.y*dZ;
	result[idx]= (abs(field_old[idx]-field_new[idx])<eps);
}
__device__ int niv_counter2;
__global__ void niv_stage_dim2(char* result){
	niv_counter2=0;
	__syncthreads();
	int dY=gridDim.x;
	int dZ=gridDim.x*blockDim.x;
	int idx=threadIdx.x+blockIdx.x*dY;
	for(int i=0;i<blockDim.x;i++)
		result[idx]=(result[idx]&&result[idx+i*dZ]);
	//atomicAdd(&niv_counter2,(int)result[idx]);
}
__device__ int niv_counter3;
__global__ void niv_stage_final(char*result){
	niv_counter3=0;
	int dY=gridDim.x;
	int idx=blockIdx.x;
	for(int i=0;i<blockDim.x;i++)
		result[idx]=(result[idx]&&result[idx+i*dY]);
	//atomicAdd(&niv_counter3, (int) result[idx]);
}

//// RELATIVE CONVERGENCE
__global__ void relNivStageDim3(char* result,double* field_old,double* field_new, double eps){
	__syncthreads();
	int dY=gridDim.x;
	int dZ=gridDim.x*blockDim.x;
	int idx=threadIdx.x+blockIdx.x*dY+blockIdx.y*dZ;
	result[idx]= (double)(abs(field_old[idx]-field_new[idx])/field_old[idx]<eps);
}

__global__ void relNivStageDim2(char* result){
	niv_counter2=0;
	__syncthreads();
	int dY=gridDim.x;
	int dZ=gridDim.x*blockDim.x;
	int idx=threadIdx.x+blockIdx.x*dY;
	for(int i=0;i<blockDim.x;i++)
		result[idx]=(result[idx]&&result[idx+i*dZ]);
	//atomicAdd(&niv_counter2,(int)result[idx]);
}

__global__ void relNivStageFinal(char*result){
	niv_counter3=0;
	int dY=gridDim.x;
	int idx=blockIdx.x;
	for(int i=0;i<blockDim.x;i++)
		result[idx]=(result[idx]&&result[idx+i*dY]);
	//atomicAdd(&niv_counter3, (int) result[idx]);
}

////
//// Max Field
__global__ void maxFieldStageDim3(double* result,const int* stateIndex,const double* fiNew){
	int dY=gridDim.x+2;
	int dZ=(gridDim.x+2)*(blockDim.x+2);
	int idx=1+threadIdx.x+(1+blockIdx.x)*dY+(1+blockIdx.y)*dZ;
	result[idx]=0;
	if(stateIndex[idx]<=57 && stateIndex[idx]>=31){

		for(int i=-1;i<2;i+=2){
			for(int j=-1; j<2; j+=2){
				for(int k=-1; k<2; k+=2){
					if(fiNew[idx]-fiNew[idx+i+j*dY+k*dZ]>result[idx]){
						result[idx]=fiNew[idx]-fiNew[idx+i+j*dY+k*dZ];
					}
				}
			}
		}
	}
}

__global__ void maxFieldStageDim2(double* result){
	int dY=(gridDim.x+2);
	int dZ=(gridDim.x+2)*(blockDim.x+2);
	int idx=1+threadIdx.x+(1+blockIdx.x)*dY;
	for(int i=0;i<blockDim.x;i++){
		if(result[idx+i*dZ]>result[idx]){
			result[idx]=result[idx+i*dZ];
		}
	}
	//atomicAdd(&niv_counter2,(int)result[idx]);
}

__global__ void maxFieldStageDim1(double*result){
	int dY=blockDim.x+2;
	int idx=1+threadIdx.x;
	for(int i=0;i<blockDim.x;i++){
		if(result[idx+i*dY]>result[idx]){
			result[idx]=result[idx+i*dY];
		}
	}
}


////
void cuLaplas::errReport(char* s, hipError_t e){
	fprintf(stderr,s,e);
}

///������������� ��������
cuLaplas::cuLaplas(int _size,int z,int limit,BORDER_CONDITION bc){
	///������ �������
	_grSz=_size;
	_slc_z=z;
	dY=_grSz;
	dZ=_grSz*_grSz;
	///���������� ��������
	_iteration_current=0;
	_iteration_total=0;
	_iteration_series=30;
	_iteration_limit=limit;
	///������� ���� �������
	/*setBC(bc);*/
	errorsF= fopen("errlog.txt","w");
	///������� ������
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(errorsF, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		//goto Error;
		return;
	}
	// Allocate GPU buffers()    .
	/////////////////////////////////////////////////////////////////////////////
	cudaStatus = hipMalloc((void**)&dev_fi, _size *_size*_size* sizeof(double));
	cudaStatus = hipMalloc((void**)&dev_niv_checks, _size *_size*_size* sizeof(char));
	cudaStatus = hipMalloc((void**)&relDev_niv_checks, _size *_size*_size* sizeof(char));
	cudaStatus = hipMalloc((void**)&dev_fi_old, _size *_size*_size* sizeof(double));
	cudaStatus = hipMalloc((void**)&dev_fi_slice, _size *_size* sizeof(double));
	//cudaStatus = hipMalloc((void**)&devQ,_size*_size*_size*(sizeof(double)));
	//cudaStatus = hipMalloc((void**)&devQOld,_size*_size*_size*(sizeof(double)));
	////////////////////////////////////////////////////////////////////////////////

	////�������� �������
	////////////////////////////////////////////////////////////////////////////////

	cudaStatus=hipMemset (dev_fi,0,sizeof(double)*_size*_size*_size);
	setDouble2
		<<<dim3(_size,1,1),dim3(_size,_size,1)>>>  (dev_fi,0.4);
	///////////////////////////////////
	cudaStatus=hipMemset (dev_fi_old,0,sizeof(double)*_size*_size*_size);
	setDouble2
		<<<dim3(_size,1,1),dim3(_size,_size,1)>>>  (dev_fi_old,0.4);
	// Check for any errors
	//////////////////////////////////


	cudaStatus=hipMemset (dev_fi_slice,0,sizeof(double)* _size * _size);
	cudaStatus=hipMemset (dev_niv_checks,0,sizeof(char)* _size * _size*_size);
	cudaStatus=hipMemset (relDev_niv_checks,0,sizeof(char)* _size * _size*_size);
	//cudaStatus=hipMemset (devQ,0,sizeof(float)*_size*_size*_size);
	//cudaStatus=hipMemset (devQOld,0,sizeof(float)*_size*_size*_size);
	//setDouble1<<<1,dim3(_size,_size,1)>>>(dev_fi_slice,0);
	//setDouble2<<<_size,dim3(_size,_size,1)>>>(dev_fi,0.4);
	//setDouble2<<<_size,dim3(_size,_size,1)>>>(dev_fi_old,0.4);
	cudaStatus=_neiman_init(dev_fi_old);
	cudaStatus=_neiman_init(dev_fi);
	//////////////////////////////////////////////////////////////////////////////////	
}

///���� ����
hipError_t cuLaplas::cpySlice(double* host_target){
	sliceKernel<<<_grSz,_grSz>>>(dev_fi_old,dev_fi_slice,_slc_z);
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.

	cudaStatus = hipDeviceSynchronize();
	cudaStatus = hipMemcpy(host_target, dev_fi_slice, _grSz *_grSz* sizeof(double), hipMemcpyDeviceToHost);
	return cudaStatus;

}
hipError_t cuLaplas::_neiman_noflow_Border(double* target,int current){

	update_border<<<dim3((_grSz-2)/32,1,(_grSz-2)),dim3(4,4,2)>>>(target);
	cudaStatus=hipGetLastError();
	cudaStatus=hipDeviceSynchronize();

	edge_update<<<1,_grSz/2>>>(target, _grSz/2+_grSz*_grSz/2, _grSz*_grSz);
	cudaStatus=hipGetLastError();
	cudaStatus=hipDeviceSynchronize();
	return cudaStatus;
}
hipError_t cuLaplas::_neiman_init(double* target){
	//////////
	yx_Borders<<<_grSz,_grSz>>>(target);
	cudaStatus=hipGetLastError();
	cudaStatus=hipDeviceSynchronize();
	/////////
	update_border<<<dim3((_grSz-2)/32,1,_grSz-2),dim3(4,4,2)>>>(target);
	cudaStatus=hipGetLastError();
	cudaStatus=hipDeviceSynchronize();
	///////////
	edge_update<<<1,_grSz/2>>>(target, _grSz/2+_grSz*_grSz/2, _grSz*_grSz);
	cudaStatus=hipGetLastError();
	cudaStatus=hipDeviceSynchronize();
	/////////
	return cudaStatus;
}
hipError_t cuLaplas::iteration(void* str_str,char* epsilon_check,double eps,float* time){

	float lapTime=0;/*
					hipEvent_t start;
					hipEvent_t stop;*/
	strmr_strct* a= (strmr_strct*)str_str;

	static int current=0;
	//hipEventCreate(&stop);
	//hipEventCreate(&start);
	//hipEventRecord(start,0);
	laplasKernel<<<dim3((_grSz-2)/32,_grSz-2,_grSz-2),dim3(4,4,2)>>>(dev_fi_old,dev_fi);
	cudaStatus=hipGetLastError();
	cudaStatus=hipDeviceSynchronize();

	//state_field_update<<<dim3(_grSz-2,_grSz-2,1),dim3(_grSz-2,1,1)>>>(a->_states,dev_fi);
	//cudaStatus=hipDeviceSynchronize();

	cudaStatus=_neiman_noflow_Border(dev_fi,current);
	cudaStatus=hipDeviceSynchronize();
	//laplasKernel<<<dim3(_grSz-2,1,1),dim3(_grSz-2,_grSz-2,1)>>>(dev_fi,dev_fi_old/*,dev_sgm,dev_q*/);
	//cudaStatus=hipGetLastError();


	//cudaStatus=_neiman_noflow_Border(dev_fi_old,current);
	//hipEventRecord(stop,0);
	//hipEventSynchronize(stop);
	//hipEventElapsedTime(&lapTime,start,stop);
	*time+=lapTime;
	current++;
	_iteration_total++;
	if (current>=_iteration_series){
		current=0;
		_iteration_current++;
	}
	swapFi();
	return cudaStatus;
}
void cuLaplas::convergence(char* epsilon_check, double eps){
	niv_stage_dim3<<<dim3(_grSz,_grSz,1),dim3(_grSz,1,1)>>>(dev_niv_checks,dev_fi_old,dev_fi,eps);
	cudaStatus=hipDeviceSynchronize();

	niv_stage_dim2<<<dim3(_grSz,1,1),dim3(_grSz,1,1)>>>(dev_niv_checks);
	cudaStatus=hipDeviceSynchronize();

	niv_stage_final<<<dim3(_grSz,1,1),1>>>(dev_niv_checks);
	cudaStatus=hipDeviceSynchronize();

	hipMemcpy(epsilon_check,dev_niv_checks,sizeof(char)*_grSz,hipMemcpyDeviceToHost);
}
void cuLaplas::RelConvergence(char* epsilon_check, double eps){
	relNivStageDim3<<<dim3(_grSz,_grSz,1),dim3(_grSz,1,1)>>>(relDev_niv_checks,dev_fi_old,dev_fi,eps);
	cudaStatus=hipDeviceSynchronize();

	relNivStageDim2<<<dim3(_grSz,1,1),dim3(_grSz,1,1)>>>(relDev_niv_checks);
	cudaStatus=hipDeviceSynchronize();

	relNivStageFinal<<<dim3(_grSz,1,1),1>>>(relDev_niv_checks);
	cudaStatus=hipDeviceSynchronize();

	hipMemcpy(epsilon_check,relDev_niv_checks,sizeof(char)*_grSz,hipMemcpyDeviceToHost);
}
cuLaplas:: ~cuLaplas(){
	hipFree(dev_fi);
	hipFree(dev_fi_old);
	hipFree(dev_fi_slice);
}
void cuLaplas::MaxSearch(const int* StateIdx){
	double* result;
	hipMalloc((void**)&result,sizeof(double)*_grSz*_grSz*_grSz);
	hipMemset(result,0,sizeof(double)*_grSz*_grSz*_grSz);
	maxFieldStageDim3<<<dim3(_grSz-2,_grSz-2,1),dim3(_grSz-2,1,1)>>>(result,StateIdx,dev_fi);
	hipDeviceSynchronize();

	maxFieldStageDim2<<<dim3(_grSz-2,1,1),dim3(_grSz-2,1,1)>>>(result);
	hipDeviceSynchronize();

	maxFieldStageDim1<<<dim3(1,1,1),dim3(_grSz-2,1,1)>>>(result);
	hipDeviceSynchronize();


	double* r= new double[_grSz];
	hipMemcpy(r,result,sizeof(double)*_grSz,hipMemcpyDeviceToHost);

	for(int i=0;i<_grSz-2;i++){
		if(r[1+i]>r[1]){
			r[1]=r[1+i];
		}
	}

	std::ofstream fileOut;
	fileOut.precision(16);
	fileOut.open("E_ot_t.txt",std::ofstream::app);
	fileOut<<r[1]<<'\n';
	fileOut.close();
	hipFree(result);

}

void cuLaplas::GetFiCentral(double*result){
	double* resTmp;
	hipMalloc((void**)&resTmp , sizeof(double)*_grSz);
	hipMemset(resTmp,0,_grSz);
	devGetFiCentral<<<dim3((_grSz-2)/32,1,1),dim3(4,4,2)>>>(dev_fi,resTmp,_grSz/2+_grSz*_grSz/2,_grSz*_grSz);
	hipDeviceSynchronize();
	hipMemcpy(result,resTmp,sizeof(double)*_grSz,hipMemcpyDeviceToHost);
	hipFree(resTmp);
}

///////////////////////////////////////////////////////////////////
///////////������ ���� ��� ��������� � ������ ���������� �������///
///////////////////////////////////////////////////////////////////

strmr_strct::strmr_strct(cuLaplas* parent){
	_size= parent->getSize();
	_slc_z=_size/2;
	hipMalloc((void**)&_states,sizeof(int)*_size*_size*_size);
	hipMemset (_states,0,sizeof(int)*_size*_size*_size);
	hipMalloc((void**)&dev_slice,sizeof(int)*_size*_size);
	hipMemset (dev_slice,0,sizeof(int)*_size*_size);
	hipMalloc((void**)&rand_results,sizeof(float)*_size*_size*_size);
	initStates
		<<<dim3(_size-2,_size-2,1), dim3(_size-2,1,1)>>>
		(_states);

	edgeInitStruct
		<<<1,_size/2 -1>>>
		(_states, _size/2+_size*_size/2, _size*_size);

	hiprandCreateGenerator(&gen, 
		HIPRAND_RNG_PSEUDO_DEFAULT);
}
//__device__ int gr_check(int* states,int id){
//	if( ) return 0;
//	return 1;
//}
hipError_t strmr_strct::cpySlice(int* host_target){
	sliceKernel_int<<<_size,_size>>>(_states,dev_slice,_slc_z);
	// Check for any errors launching the kernel
	/*cudaStatus = hipGetLastError();*/
	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.

	/*cudaStatus = */hipDeviceSynchronize();
	/*cudaStatus = */hipMemcpy(host_target, dev_slice, _size *_size* sizeof(int), hipMemcpyDeviceToHost);
	return hipSuccess;

}
__global__ void initStates(int* A){
	int dY=gridDim.x+2;
	int dZ=(gridDim.x+2)*(gridDim.y+2);
	int i0=1+blockIdx.x +(blockIdx.y +1)*dY+(threadIdx.x +1)*dZ;
	A[i0]=200;

}

__global__ void edgeInitStruct(int*A,int xy_idx,int dZ){
	int i0=xy_idx+(1+threadIdx.x)*dZ;
	A[i0]=100;
	if ((threadIdx.x+1)==blockDim.x){
		A[i0]=101;
	}
}

///���������  ��������
__device__ int d_count_200_true=0;
__device__ int d_count_101_true=0;
__device__ int d_count_0_30_true=0;
__device__ int d_count_31_60_true=0;
__device__ int d_count_calls=0;
__device__ int d_count_passed_checks[2];

__device__ void checkStructure(float rand_val,double* field,int* states,
							   int dY,int dZ,int id_to)
{
	if	(states[id_to]>=61){
		if	(states[id_to]<=87){
			//atomicAdd(&d_count_31_60_true, 1);
			int ii=((states[id_to]-61)%9)%3-1;
			int jj=((states[id_to]-61)%9)/3-1;
			int kk=(states[id_to]-61)/9 -1;
			int id_from1 =id_to+ii+jj*dY+kk*dZ;
			if(states[id_from1]<57 && states[id_from1]>=31){
				states[id_from1]-=30;
			}
			//printf("%d %d %d \n",ii,jj,kk);
			states[id_to]-=30;
			//atomicAdd(&d_count_31_60_true, 1);
		}
	}

	if(states[id_to]==200 ){
		int tmpState;
		int tmp_from;
		int check =0;
		int tmpDiag;
		double tmpE=0;
		//atomicAdd(&d_count_200_true, 1);
		for (int i=-1;i<2;i++){
			for(int j=-1;j<2;j++){
				for(int k=-1;k<2;k++){

					//printf("%d %d %d \n",i,j,k);
					int	id_from;
					id_from=id_to+i+j*dY+k*dZ;
					if((i!=0)||(j!=0)||(k!=0)){
						if( states[id_from]==101 || ((states[id_from]<=57) &&(states[id_from]>0)))
						{
							//atomicAdd(&d_count_101_true, 1);
							double randomized_field =0;
							randomized_field=abs(field[id_to]-field[id_from])/sqrt((double)i*i+j*j+k*k)-log(rand_val)*0.2;
							if (randomized_field>tmpE){   
								if(randomized_field>0.9){	
									check =1;
									tmp_from=id_from;
									bool A=(i!=0);
									bool B=(j!=0);
									bool C=(k!=0);
									tmpDiag= (int) !(A^B^C) || (A&&B&&C);
									tmpE=randomized_field;
									tmpState= 30*tmpDiag +31+ (1+i)+(1+j)*3+(1+k)*9;
									//atomicAdd(&d_count_passed_checks[0], 1);
								}

							}
						}
					}
				}
			}
		}

		if (check){
			if( (states[tmp_from]<=57) &&(states[tmp_from]>30))
			{
				atomicSub(states+tmp_from, 30*tmpDiag);
			}
			states[id_to]=tmpState;
		}
	}
	/*return 0;*/
}

__global__ void gr_iterate(int* states, double* field,float* uniformrand){
	d_count_200_true=0;
	d_count_101_true=0;
	d_count_0_30_true=0;
	d_count_31_60_true=0;
	int dY=blockDim.x*blockDim.y*gridDim.x+2;
	int dZ=dY*(gridDim.y+2);
	int id=(1+ threadIdx.x +threadIdx.y*4+blockIdx.x*16)+(1+blockIdx.y)*dY +(blockIdx.z +1)*dZ;
	/*atomicAdd(&d_count_calls, 1);*/
	checkStructure(uniformrand[id],field,states,
		dY,dZ,id);

	__syncthreads();

}


__global__ void d_count_report(){
	printf("d_count_calls: %d  \n",d_count_calls);
	printf("d_count_200_true: %d  \n",d_count_200_true);
	printf("d_count_101_true: %d  \n",d_count_101_true);
	printf("d_count_31_60_true: %d  \n",d_count_31_60_true);
	printf("d_count_passed_checks[0]: %d  \n",d_count_passed_checks[0]);
	printf("d_count_passed_checks[1]: %d  \n",d_count_passed_checks[1]);
	//printf("niv2: %d  \n",niv_counter2);
	//printf("niv3: %d  \n",niv_counter3);

}
int h_N_grow_counter=0;
void strmr_strct::count_report(/*std::fstream* a,*/double b){
	d_count_report<<<1,1>>>();
	hipDeviceSynchronize();
	printf("neodnorodnost: %f  \n",b);
}
__global__ void sliceKernel_int(int* A,int*result,const int z){
	int dY=blockDim.x;
	int dZ=(gridDim.x)*(blockDim.x);

	int idx=threadIdx.x+blockIdx.x*dZ;

	int a;
	for(int i=0;i<gridDim.x;i++){
		a=A[idx+i*dY];
		if((a!=200)&&(a!=0))break;
	}
	result[threadIdx.x+blockIdx.x*dY]=a;
}
__global__ void StructureIsDelayed(int*states){
	int idTo=1+threadIdx.x+threadIdx.y*4+blockIdx.x*16+(blockIdx.y +1)*(gridDim.x*blockDim.x*blockDim.y+2)+(1+blockIdx.z)*(gridDim.x*blockDim.x*blockDim.y+2)*(gridDim.y+2);
	if((states[idTo]<=60) &&(states[idTo]>=31)){
		states[idTo]-=30;
	}
}
void strmr_strct::cu_iterate(double* field){
	hiprandSetPseudoRandomGeneratorSeed(gen,seed_rng());
	hiprandGenerateUniform(gen,rand_results,(_size)*(_size)*(_size));
	hipDeviceSynchronize();
	gr_iterate<<<dim3((_size-2)/16,_size-2,_size-2),dim3(4,4,1)>>>
		(_states, field, rand_results);
	hipDeviceSynchronize();
	//StructureIsDelayed<<<dim3((_size-2)/16,_size-2,_size-2),dim3(4,4,1)>>>
	//	(_states);
	hipDeviceSynchronize();
}

//__device__ int StructureIsGrowing(